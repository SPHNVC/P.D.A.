#include "hip/hip_runtime.h"
#include "filter.cuh"


using namespace std;



__global__ void kernel_sobel_filter(const uchar * device_input_data, uchar * device_output_data, const uint height, const uint width) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	/* Bound check */
	if (x < 0 || x > width || y > height || y < 0)
		return;

	/* To detect horizontal lines. This is effectively the dx. */
	const int sobel_x[3][3] = {
		{ -1, 0, 1 },
		{ -2, 0, 2 },
		{ -1, 0, 1 }
	};
	/* To detect vertical lines. This is effectively the dy. */
	const int sobel_y[3][3] = {
		{ -1, -2, -1 },
		{ 0,   0,  0 },
		{ 1,   2,  1 }
	};

	double magnitude_x = 0;
	double magnitude_y = 0;

	for (uint j = 0; j < 3; ++j) {
		for (uint i = 0; i < 3; ++i) {
			const int x_focus = i + x;
			const int y_focus = j + y;
			const int index = y_focus + x_focus * width;
			magnitude_x += device_input_data[index] * sobel_x[i][j];
			magnitude_y += device_input_data[index] * sobel_y[i][j];
		}
	}
	double magnitude = sqrt(magnitude_x * magnitude_x + magnitude_y * magnitude_y);

	/* Edge cases of MIN or MAX RGB after the Sobel operator is applied */
	if (magnitude < MIN_RGB_VALUE)
		magnitude = MIN_RGB_VALUE;
	if (magnitude > MAX_RGB_VALUE)
		magnitude = MAX_RGB_VALUE;

	device_output_data[y + x * width] = magnitude;
}

/**
* Wrapper for calling the kernel.
*/
double Filter::sobel_filter_gpu(const uchar * host_data, uchar * output, const uint height, const uint width) {
	const int size = height * width * sizeof(uchar);

	/* Allocate device memory for the result. */
	/* Note that output to hold the HOST memory has already been allocated for. */
	void * device_input_data = nullptr;
	void * device_output_data = nullptr;

	if (hipMalloc((void **)& device_input_data, size) != hipSuccess)
		std::cerr << get_cuda_error() << std::endl;

	if (hipMalloc((void **)& device_output_data, size) != hipSuccess)
		std::cerr << get_cuda_error() << std::endl;

	/* Copy the input data to the device. */
	if (hipMemcpy(device_input_data, host_data, size, hipMemcpyHostToDevice) != hipSuccess)
		std::cerr << get_cuda_error() << std::endl;

	/* Launch the kernel! */
	dim3 grid(GRID_X, GRID_Y, 1);
	dim3 block(EXPECTED_WIDTH / GRID_X, EXPECTED_HEIGHT / GRID_Y, 1);

	kernel_sobel_filter << <grid, block >> >((uchar*)device_input_data, (uchar*)device_output_data, height, width);

	if (hipMemcpy(output, device_output_data, size, hipMemcpyDeviceToHost) != hipSuccess)
		std::cerr << get_cuda_error() << std::endl;

	hipFree(device_input_data);
	hipFree(device_output_data);

	/* Capture the device copy-compute-copy time. */
	return get_timer_value();
}

void Filter::sobel_filter_cpu(const uchar * input, uchar * output, const uint height, const uint width) {
	/* To detect horizontal lines. This is effectively the dx. */
	const int sobel_x[3][3] = {
		{ -1, 0, 1 },
		{ -2, 0, 2 },
		{ -1, 0, 1 }
	};
	/* To detect vertical lines. This is effectively the dy. */
	const int sobel_y[3][3] = {
		{ -1, -2, -1 },
		{ 0,   0,  0 },
		{ 1,   2,  1 }
	};

	/**
	* Iterate and perform Sobel Filter on every pixel.
	* Make outer loop the ys so that successive reads are as close to each other as possible,
	* i.e. for single-threaded CPU code, it is most important for caching, but for GPUs it is
	* most important for coalesced memory access (and maybe caching).
	* If we iterate over the rows first, we have 0 coalescing then.
	*/
	for (uint y = 1; y < height - 2; ++y) {
		for (uint x = 1; x < width - 2; ++x) {
			const double pixel_x = (double)(
				(sobel_x[0][0] * input[get_array_index(x - 1, y - 1, width)]) +
				(sobel_x[0][1] * input[get_array_index(x, y - 1, width)]) +
				(sobel_x[0][2] * input[get_array_index(x + 1, y - 1, width)]) +
				(sobel_x[1][0] * input[get_array_index(x - 1, y, width)]) +
				(sobel_x[1][1] * input[get_array_index(x, y, width)]) +
				(sobel_x[1][2] * input[get_array_index(x + 1, y, width)]) +
				(sobel_x[2][0] * input[get_array_index(x - 1, y + 1, width)]) +
				(sobel_x[2][1] * input[get_array_index(x, y + 1, width)]) +
				(sobel_x[2][2] * input[get_array_index(x + 1, y + 1, width)])
				);
			const double pixel_y = (double)(
				(sobel_y[0][0] * input[get_array_index(x - 1, y - 1, width)]) +
				(sobel_y[0][1] * input[get_array_index(x, y - 1, width)]) +
				(sobel_y[0][2] * input[get_array_index(x + 1, y - 1, width)]) +
				(sobel_y[1][0] * input[get_array_index(x - 1, y, width)]) +
				(sobel_y[1][1] * input[get_array_index(x, y, width)]) +
				(sobel_y[1][2] * input[get_array_index(x + 1, y, width)]) +
				(sobel_y[2][0] * input[get_array_index(x - 1, y + 1, width)]) +
				(sobel_y[2][1] * input[get_array_index(x, y + 1, width)]) +
				(sobel_y[2][2] * input[get_array_index(x + 1, y + 1, width)])
				);

			/* Pythagorean Theorem for the magnitude. Push down index into 1D array.*/
			double magnitude = sqrt(pixel_x * pixel_x + pixel_y * pixel_y);

			/* Edge cases of MIN or MAX RGB after the Sobel operator is applied */
			if (magnitude < MIN_RGB_VALUE)
				magnitude = MIN_RGB_VALUE;
			if (magnitude > MAX_RGB_VALUE)
				magnitude = MAX_RGB_VALUE;

			output[y + x * width] = magnitude;
		}
	}
}

double Filter::sobel_filter_verify_errors(const uchar * input_data, const uchar * compare, const uint height, const uint width) {
	uchar * cpu_results = (uchar *)malloc(height * width * sizeof(uchar));
	if (!cpu_results) {
		throw runtime_error("Problems in reserving memory for the CPU version.");
	}

	/* Do the Sobel Filter using the CPU. */
	sobel_filter_cpu(input_data, cpu_results, height, width);

	const char * cpu_saved_file = "cpu_output.pgm";

	if (!sdkSavePGM<uchar>(cpu_saved_file, cpu_results, width, height)) {
		throw runtime_error("Error in saving the output image!");
	}
	cout << "Using the CPU version, we saved the image with filename: " << cpu_saved_file << endl;

	/* Walk through and compare the pixels of the images to see how many are wrong. */
	uint error_pixel_count = 0;
	for (uint i = 0; i < height * width; ++i) {
		if (cpu_results[i] != compare[i])
			++error_pixel_count;
	}

	/* Return the percentage of how many pixels are wrong. */
	return error_pixel_count / (height * width);
}
